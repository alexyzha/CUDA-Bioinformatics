
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
    printf("B: %u, T: %u\n",blockIdx.x,threadIdx.x);
}

int main(int argc, char* argv[]) {

    printf("INIT\n");
    
    hello<<<7,7>>>();
    
    hipError_t err = hipDeviceSynchronize();
    if(err != hipSuccess) {
        fprintf(stderr, "CUDA ERR: %s\n", hipGetErrorString(err));
        return -1;
    }

    return 0;
}