#include "hip/hip_runtime.h"
#include "headers/test_util.cuh"
#include "headers/global_wrappers.cuh"
#include "headers/test_prototypes.cuh"
#include "../../cpu/headers/fx_util.h"

void CU_WRAPPER_TESTS(std::vector<TEST_RESULT*>& RESULTS) {
    srand(0xDEADBEEF);

    /*
     *  Phred: ! = min, K = max, 6/7 ~= mid
     */

    // Filter fastq by average discard whole
    RESULTS.push_back(TEST("CU_WRAPPER_TESTS", "FILTER_FQ_AVERAGE_DISCARD_WHOLE", [](){
        // Create test variables
        std::vector<fq_read*> reads;
        std::unordered_set<std::string> good_ids = {"10"};
        char THRESH = '6';
        for(int i = 0; i < 10; ++i) {
            std::string qual = "";
            std::string seq = "";
            int sum = 0;
            for(int i = 0; i < 100; ++i) {
                char q = (rand() % 42) + 33;
                sum += q;
                qual.push_back(q);
                seq.push_back('*');
            }

            // Check by id
            if(sum >= (THRESH * 100)) {
                good_ids.insert(std::to_string(i));
            }
            reads.push_back(new fq_read(
                std::to_string(i),
                100,
                seq,
                qual
            ));
        }

        // Add one guaranteed to pass
        reads.push_back(new fq_read(
            "10",
            100,
            std::string(100, 'A'),
            std::string(100, '8')
        ));

        // Run kernel wrapper
        std::vector<fq_read*> ret = cu_filter_fq(reads, AVERAGE_DISCARD_WHOLE, THRESH, 0, 0.0);

        // Check results against good_ids
        for(auto& read : ret) {
            EXPECT_TRUE(good_ids.count(read->get_id()));
            good_ids.erase(read->get_id());
        }
        EXPECT_TRUE(good_ids.empty());

        // Clean
        for(auto& read : reads) {
            delete read;
        }
    }));

    // Filter fastq by single discard whole
    RESULTS.push_back(TEST("CU_WRAPPER_TESTS", "FILTER_FQ_SINGLE_DISCARD_WHOLE", [](){
        std::vector<fq_read*> reads;
        std::unordered_set<std::string> good_ids = {"10"};
        char THRESH = '7';
        for(int i = 0; i < 10; ++i) {
            std::string qual = "";
            std::string seq = "";
            char min = 255;
            for(int i = 0; i < 100; ++i) {
                char q = (rand() % 42) + 33;
                min = (min < q ? min : q);
                qual.push_back(q);
                seq.push_back('*');
            }

            // Check by id
            if(min >= THRESH) {
                good_ids.insert(std::to_string(i));
            }
            reads.push_back(new fq_read(
                std::to_string(i),
                100,
                seq,
                qual
            ));
        }

        // Add one guaranteed to pass
        reads.push_back(new fq_read(
            "10",
            100,
            std::string(100, 'A'),
            std::string(100, '8')
        ));

        // Run kernel wrapper
        std::vector<fq_read*> ret = cu_filter_fq(reads, SINGLE_DISCARD_WHOLE, THRESH, 0, 0.0);

        // Check results against good_ids
        for(auto& read : ret) {
            EXPECT_TRUE(good_ids.count(read->get_id()));
            good_ids.erase(read->get_id());
        }
        EXPECT_TRUE(good_ids.empty());

        // Clean
        for(auto& read : reads) {
            delete read;
        }
    }));

    // Trim fastq by sliding window - trim at first window where avg(window) < thresh
    RESULTS.push_back(TEST("CU_WRAPPER_TESTS", "FILTER_FQ_SLIDING_WINDOW", [](){
        std::vector<fq_read*> reads;
        char THRESH = '6';
        std::vector<std::string> exp_seq = {"ABCDEFG", "ABCDE", "ABCDEFGHIJ"};
        std::vector<std::string> exp_qual = {"6666666", "77556", "7777777777"};

        // Add one with very obvious cut
        reads.push_back(new fq_read(
            "0",
            10,
            "ABCDEFGHIJ",
            "6666666555" 
        ));

        // Add one with less obvious cut
        reads.push_back(new fq_read(
            "1",
            10,
            "ABCDEFGHIJ",
            "7755667777"
        ));

        // Add one with no cut
        reads.push_back(new fq_read(
            "2",
            10,
            "ABCDEFGHIJ",
            "7777777777"
        ));

        // Run kernel wrapper
        std::vector<fq_read*> ret = cu_filter_fq(reads, SLIDING_WINDOW, THRESH, 5, 0.0);

        // Check results
        EXPECT_EQ(ret.size(), 3);
        for(int i = 0; i < 3; ++i) {
            EXPECT_EQ(ret[i]->get_seq(), exp_seq[i]);
            EXPECT_EQ(ret[i]->get_quality(), exp_qual[i]);
        }

        // Clean
        for(auto& read : reads) {
            delete read;
        }
    }));

    // Filter fastq by proportion discard whole
    RESULTS.push_back(TEST("CU_WRAPPER_TESTS", "FILTER_FQ_PROPORTION_DISCARD_WHOLE", [](){
        std::vector<fq_read*> reads;
        char THRESH = '6';
        
        // Add one guaranteed to fail
        reads.push_back(new fq_read(
            "0",
            100,
            std::string(100, 'A'),
            std::string(100, '5')
        ));

        // Add one guaranteed to pass
        reads.push_back(new fq_read(
            "1",
            100,
            std::string(100, 'A'),
            std::string(100, '7')
        ));

        // Run kernel wrapper
        std::vector<fq_read*> ret = cu_filter_fq(reads, PROPORTION_DISCARD_WHOLE, THRESH, 0, 0.5);

        // Check results
        EXPECT_EQ(ret.size(), 1);
        EXPECT_EQ(ret[0]->get_id(), "1");

        // Clean
        for(auto& read : reads) {
            delete read;
        }
    }));

    // Count kmer kernel test
    RESULTS.push_back(TEST("CU_WRAPPER_TESTS", "COUNT_KMERS", [](){
        // Create reads
        std::vector<fq_read*> reads;
        char bases[4] = {'A', 'C', 'G', 'T'};
        for(int i = 0; i < 100; ++i) {
            std::string seq(1000, '\0');
            std::string qual(1000, '?');
            for(int j = 0; j < 1000; ++j) {
                seq[j] = bases[rand() % 4];
            }
            reads.push_back(new fq_read(
                std::to_string(i),
                1000,
                seq,
                qual
            ));
        }

        // Expected from cpu code
        std::unordered_map<uint64_t, uint64_t> exp = count_kmer(reads, 7);

        // Run kernel
        std::unordered_map<uint64_t, uint64_t> ret = cu_count_kmers(reads, 7);

        // Check results
        EXPECT_EQ(ret.size(), exp.size());
        for(auto& [key, value] : ret) {
            EXPECT_NE(exp.find(key), exp.end());
            EXPECT_EQ(exp[key], value);
        }

        // Clean
        for(auto& read : reads) {
            delete read;
        }
    }));

    // Index kmer kernel test
    RESULTS.push_back(TEST("CU_WRAPPER_TESTS", "INDEX_KMERS", [](){
        // Create testing variables
        std::vector<fq_read*> reads;
        std::vector<std::string> seqs = {"ACG", "GAT", "CAT", "CAG", "ACG"};
        std::unordered_map<uint64_t, std::unordered_set<int>> exp = {
            {0b0001, {0, 4}}, {0b0110, {0, 4}}, {0b1000, {1}},
            {0b0011, {1, 2}}, {0b0100, {2, 3}}, {0b0010, {3}}
        };
        for(int i = 0; i < 5; ++i) {
            reads.push_back(new fq_read(
                std::to_string(i),
                3,
                seqs[i],
                "???"
            ));
        }

        // Run kernel
        std::unordered_map<uint64_t, std::unordered_set<int>> ret = cu_index_kmers(reads, 2);

        // Check results
        EXPECT_EQ(ret.size(), exp.size());
        for(auto& [key, value] : ret) {
            EXPECT_NE(exp.find(key), exp.end());
            EXPECT_EQ(exp[key].size(), value.size());
            for(auto& index : value) {
                EXPECT_TRUE(exp[key].count(index));
            }
        }

        // Clean
        for(auto& read : reads) {
            delete read;
        }
    }));

}