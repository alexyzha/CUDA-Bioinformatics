#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ hello() {
    printf("B: %u, T: %u\n",blockIdx.x,threadIdx.x);
}

int main(int argc, char argv[]) {

    hello<<<7,7>>>();
    hipDeviceSynchronize();

    return 0;
}