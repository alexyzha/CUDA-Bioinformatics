#include "headers/wrappers.cuh"

std::vector<fq_read*> cu_filter_fq(const std::vector<fq_read*>& READS, char FILTER_MODE, char THRESH, size_t K, double PROPORTION) {
    // Size variables
    int READ_LEN = READS.size();
    int FILTER_LENGTH = (READ_LEN / 64) + 1;
    int THREADS = MAX_THREADS;
    int BLOCKS = (READ_LEN / THREADS) + 1;
    
    // Host variables
    std::string temp = "";
    std::vector<uint32_t> h_offsets(READ_LEN);
    for(int i = 0; i < READ_LEN; ++i) {
        temp += READS[i]->get_quality();
        h_offsets[i] = temp.size() - 1;
    }
    int SEQ_LEN = temp.size() + 1;                  // temp + '\0'
    const char* h_allseq = temp.c_str();
    
    // Device variables
    char* d_allseq;
    uint32_t* d_offsets;    
    uint64_t* d_filter;

    // Allocate mem for device variables
    CUDA_CHECK(hipMalloc(&d_allseq, sizeof(char) * SEQ_LEN));
    CUDA_CHECK(hipMalloc(&d_offsets, sizeof(uint32_t) * READ_LEN));
    CUDA_CHECK(hipMalloc(&d_filter, sizeof(uint64_t) * FILTER_LENGTH));
    
    // Copy mem host -> device/set mem
    CUDA_CHECK(hipMemset(d_filter, 0, sizeof(uint64_t) * FILTER_LENGTH));
    CUDA_CHECK(hipMemcpy(d_allseq, h_allseq, sizeof(char) * SEQ_LEN, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, h_offsets.data(), sizeof(uint32_t) * READ_LEN, hipMemcpyHostToDevice));

    // Run kernels
    if(FILTER_MODE == SLIDING_WINDOW) {
        cu_filter_reads_sw <<<BLOCKS, THREADS>>> (
            d_allseq, 
            d_offsets, 
            READ_LEN, 
            K, 
            THRESH, 
            PROPORTION
        );
    } else {
        cu_filter_reads <<<BLOCKS, THREADS>>> (
            d_allseq, 
            d_offsets, 
            READ_LEN, 
            FILTER_MODE, 
            THRESH, 
            d_filter, 
            PROPORTION
        );
    }
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy mem back
    std::vector<fq_read*> filtered_reads;
    if(FILTER_MODE != SLIDING_WINDOW) {
        // Copy mem back
        std::vector<uint64_t> ret_filter(FILTER_LENGTH, 0);
        CUDA_CHECK(hipMemcpy(ret_filter.data(), d_filter, sizeof(uint64_t) * FILTER_LENGTH, hipMemcpyDeviceToHost));

        // Filter and return
        for(int i = 0; i < READ_LEN; ++i) {
            int f_index = i / 64;
            int f_bit = i % 64;
            if(ret_filter[f_index] & ((ULL)1 << f_bit)) {
                filtered_reads.push_back(READS[i]);
            }
        }
    } else {
        // Copy mem back
        std::vector<char> temp_allseq(SEQ_LEN);
        CUDA_CHECK(hipMemcpy(temp_allseq.data(), d_allseq, sizeof(char) * SEQ_LEN, hipMemcpyDeviceToHost));
        const char* ret_allseq = temp_allseq.data();

        // Filter, trim, and return
        int seq_begin = 0;
        for(int i = 0; i < READ_LEN; ++i) {
            int og_size = READS[i]->size();
            
            // Get seq substring from seq buffer
            std::string cur_seq(ret_allseq + seq_begin);
            if(cur_seq.size() >= og_size) {
                filtered_reads.push_back(READS[i]);
            } else if(cur_seq.size()) {
                filtered_reads.push_back(new fq_read(
                    READS[i]->get_id(),
                    cur_seq.size(),
                    READS[i]->get_seq().substr(0, cur_seq.size()),
                    cur_seq,
                    READS[i]->get_metadata()
                ));
            }

            // Move to next seq begin
            seq_begin = h_offsets[i] + 1;
        }
    }
    
    // Cleanup and return
    CUDA_CHECK(hipFree(d_filter));
    CUDA_CHECK(hipFree(d_offsets));
    CUDA_CHECK(hipFree(d_allseq));
    return filtered_reads;
}

std::unordered_map<uint64_t, uint64_t> cu_count_kmers(const std::vector<fq_read*>& READS, size_t K) {
    /*
     *  Given: 16 bytes per kh_pair, MAP_SIZE = 2 * total kmers, total bytes = 4^k * 2 * 16
     *  k = 13 requires INT_MAX bytes = 2gb vram/normal ram
     *  My GPU has 8gb vram...
     *  Note to self: consider setting K_MAX as 13 or 14
     *  -> Map needs size to be 2x the number of elements it's holding for better nocollide
     */
    if(K > 13) {
        throw std::runtime_error("K is too large, possible memory issues.");
    }
    
    // Size variables 
    int READ_LEN = READS.size(); 
    uint64_t MAP_SIZE = (1ULL << (2 * K)) * 2;
    int THREADS = MAX_THREADS;
    int BLOCKS = (READ_LEN / THREADS) + 1;
    
    // Host variables
    std::string temp = "";
    std::vector<uint32_t> h_offsets(READ_LEN);
    for(int i = 0; i < READ_LEN; ++i) {
        temp += READS[i]->get_seq();
        h_offsets[i] = temp.size() - 1;
    }
    int SEQ_LEN = temp.size() + 1;                  // temp + '\0'
    const char* h_allseq = temp.c_str();
    
    // Device variables
    char* d_allseq;
    uint32_t* d_offsets;
    kh_pair<uint64_t>* d_map = kh_construct<uint64_t>(MAP_SIZE);

    // Allocate mem for device variables
    CUDA_CHECK(hipMalloc(&d_allseq, sizeof(char) * SEQ_LEN));
    CUDA_CHECK(hipMalloc(&d_offsets, sizeof(uint32_t) * READ_LEN));

    // Copy mem host -> device
    CUDA_CHECK(hipMemcpy(d_allseq, h_allseq, sizeof(char) * SEQ_LEN, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, h_offsets.data(), sizeof(uint32_t) * READ_LEN, hipMemcpyHostToDevice));

    // Run kernel
    cu_kmer_count <<<BLOCKS, THREADS>>> (
        d_map, 
        d_allseq, 
        d_offsets, 
        K, 
        READ_LEN, 
        MAP_SIZE
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy mem back
    std::vector<kh_pair<uint64_t>> h_map(MAP_SIZE);
    CUDA_CHECK(hipMemcpy(h_map.data(), d_map, sizeof(kh_pair<uint64_t>) * MAP_SIZE, hipMemcpyDeviceToHost));

    // Format data
    std::unordered_map<uint64_t, uint64_t> ret;
    ret.reserve(MAP_SIZE / 8);
    for(int i = 0; i < MAP_SIZE; ++i) {
        if(h_map[i].key == (ULL)EMPTY) {
            continue;
        }
        ret[h_map[i].key] = h_map[i].value;
    }

    // Clean and return
    CUDA_CHECK(hipFree(d_offsets));
    CUDA_CHECK(hipFree(d_allseq));
    CUDA_CHECK(hipFree(d_map));
    return ret;
}

std::unordered_map<uint64_t, std::unordered_set<int>> cu_index_kmers(const std::vector<fq_read*>& READS, size_t K) {
    
    // Size variables
    int READ_LEN = READS.size(); 
    uint64_t MAP_SIZE = (1ULL << (2 * K)) * 2;
    int THREADS = MAX_THREADS;
    int BLOCKS = (READ_LEN / THREADS) + 1;

    // Host variables
    std::string temp = "";
    std::vector<uint32_t> h_offsets(READ_LEN);
    for(int i = 0; i < READ_LEN; ++i) {
        temp += READS[i]->get_seq();
        h_offsets[i] = temp.size() - 1;
    }
    int SEQ_LEN = temp.size() + 1;                  // temp + '\0'
    const char* h_allseq = temp.c_str();
    
    // Device variables
    char* d_allseq;
    uint32_t* d_offsets;
    kh_pair<uint32_t[MAP_MAX_INDICES + 1]>* d_map = kh_construct<uint32_t[MAP_MAX_INDICES + 1]>(MAP_SIZE);

    // Allocate mem for device variables
    CUDA_CHECK(hipMalloc(&d_allseq, sizeof(char) * SEQ_LEN));
    CUDA_CHECK(hipMalloc(&d_offsets, sizeof(uint32_t) * READ_LEN));

    // Copy mem host -> device
    CUDA_CHECK(hipMemcpy(d_allseq, h_allseq, sizeof(char) * SEQ_LEN, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, h_offsets.data(), sizeof(uint32_t) * READ_LEN, hipMemcpyHostToDevice));

    // Run kernel
    cu_kmer_index <<<BLOCKS, THREADS>>> (
        d_map,
        d_allseq,
        d_offsets,
        K,
        READ_LEN,
        MAP_SIZE
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy mem back
    std::vector<kh_pair<uint32_t[MAP_MAX_INDICES + 1]>> h_map(MAP_SIZE);
    CUDA_CHECK(hipMemcpy(h_map.data(), d_map, sizeof(kh_pair<uint32_t[MAP_MAX_INDICES + 1]>) * MAP_SIZE, hipMemcpyDeviceToHost));

    // Format data, reserve space
    std::unordered_map<uint64_t, std::unordered_set<int>> ret;
    ret.reserve(MAP_SIZE / 4);
    for(int i = 0; i < MAP_SIZE; ++i) {
        if(h_map[i].key == (ULL)EMPTY) {
            continue;
        }

        // Insert up until MAP_MAX_INDICES matching indices
        for(int j = 1; j <= (h_map[i].value[0] - EMPTY); ++j) {
            ret[h_map[i].key].insert(h_map[i].value[j]);
        }
    }

    // Clean and return
    CUDA_CHECK(hipFree(d_offsets));
    CUDA_CHECK(hipFree(d_allseq));
    CUDA_CHECK(hipFree(d_map));
    return ret;
}

std::vector<std::unordered_set<int>*> cu_cluster_by_kmer(const std::vector<fq_read*>& READS, size_t K, size_t THRESH) {
    /*
     *  Given: 130 * 8 = 1040 bytes per kh_pair, MAP_SIZE = 2 * total kmers, total bytes = 4^k * 2 * 1040
     *  k = 10 requires 2.18B bytes = 2gb vram/normal ram
     *  My GPU has 8gb vram...
     *  Note to self: consider setting K_MAX as 10 or 11
     *  -> Map needs size to be 2x the number of elements it's holding for better nocollide
     */
    if(K > 10) {
        throw std::runtime_error("K is too large, possible memory issues.");
    }

    // Size variables
    int READ_LEN = READS.size(); 
    uint64_t MAP_SIZE = (1ULL << (2 * K)) * 2;
    uint64_t MAX_EDGES = (ULL)READ_LEN * MAP_MAX_INDICES;
    uint64_t UF_SIZE = (ULL)READ_LEN;
    int THREADS = MAX_THREADS;
    int CLUSTER_MAP_SIZE = READ_LEN * 2;

    // Host variables
    std::string temp = "";
    std::vector<uint32_t> h_offsets(READ_LEN);
    for(int i = 0; i < READ_LEN; ++i) {
        temp += READS[i]->get_seq();
        h_offsets[i] = temp.size() - 1;
    }
    int SEQ_LEN = temp.size() + 1;                  // temp + '\0'
    const char* h_allseq = temp.c_str();
    
    // Device variables for kmer indexing
    char* d_allseq;
    uint32_t* d_offsets;
    kh_pair<uint32_t[MAP_MAX_INDICES + 1]>* d_map = kh_construct<uint32_t[MAP_MAX_INDICES + 1]>(MAP_SIZE);

    // Allocate mem for device variables
    CUDA_CHECK(hipMalloc(&d_allseq, sizeof(char) * SEQ_LEN));
    CUDA_CHECK(hipMalloc(&d_offsets, sizeof(uint32_t) * READ_LEN));

    // Copy mem host -> device
    CUDA_CHECK(hipMemcpy(d_allseq, h_allseq, sizeof(char) * SEQ_LEN, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, h_offsets.data(), sizeof(uint32_t) * READ_LEN, hipMemcpyHostToDevice));

    // Run kmer indexing kernel
    cu_kmer_index <<<(READ_LEN / THREADS) + 1, THREADS>>> (
        d_map,
        d_allseq,
        d_offsets,
        K,
        READ_LEN,
        MAP_SIZE
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Device variables for kmer overlaps
    uint32_t* d_edgelist;
    uint32_t* d_edgecount_ptr;

    // Allocate mem for device variables
    CUDA_CHECK(hipMalloc(&d_edgelist, sizeof(uint32_t) * MAX_EDGES));
    CUDA_CHECK(hipMalloc(&d_edgecount_ptr, sizeof(uint32_t)));

    // Set values for device variables
    CUDA_CHECK(hipMemset(d_edgelist, 0, sizeof(uint32_t) * MAX_EDGES));
    CUDA_CHECK(hipMemset(d_edgecount_ptr, 0, sizeof(uint32_t)));

    // Run kmer overlap kernel
    cu_get_kmer_overlaps <<<(MAP_SIZE / THREADS) + 1, THREADS>>> (
        d_map,
        MAP_SIZE,
        d_edgelist,
        d_edgecount_ptr,
        MAX_EDGES
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Device/host variables for union find
    cu_union_find* d_uf = cu_uf_construct(UF_SIZE);
    uint32_t h_edgecount = 0;
    CUDA_CHECK(hipMemcpy(&h_edgecount, d_edgecount_ptr, sizeof(uint32_t), hipMemcpyDeviceToHost));

    // Run UF kernel
    cu_get_uf <<<((h_edgecount / 2) / THREADS) + 1, THREADS>>> (
        d_uf,
        UF_SIZE,
        READ_LEN,
        d_edgelist,
        h_edgecount
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Device variables for cluster kernel
    kh_pair<uint32_t[MAP_MAX_INDICES + 1]>* d_clusters = kh_construct<uint32_t[MAP_MAX_INDICES + 1]>(CLUSTER_MAP_SIZE);

    // Run cluster kernel
    cu_get_clusters <<<(UF_SIZE / THREADS) + 1, THREADS>>> (
        d_uf,
        d_clusters,
        UF_SIZE,
        CLUSTER_MAP_SIZE,
        THRESH
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy mem back
    std::vector<kh_pair<uint32_t[MAP_MAX_INDICES + 1]>> h_ret_map(CLUSTER_MAP_SIZE);
    CUDA_CHECK(hipMemcpy(h_ret_map.data(), d_clusters, sizeof(kh_pair<uint32_t[MAP_MAX_INDICES + 1]>) * CLUSTER_MAP_SIZE, hipMemcpyDeviceToHost));

    // Format data
    std::vector<std::unordered_set<int>*> ret(READ_LEN, nullptr);

    // Enter data
    for(int i = 0; i < CLUSTER_MAP_SIZE; ++i) {
        if(h_ret_map[i].key == (ULL)EMPTY) {
            continue;
        }

        // Insert up until MAP_MAX_INDICES indices
        int root = h_ret_map[i].key;
        if(!ret[root]) {
            ret[root] = new std::unordered_set<int>();
        }
        for(int j = 1; j <= (h_ret_map[i].value[0] - EMPTY); ++j) {
            ret[root]->insert(h_ret_map[i].value[j]);
        }
    }

    // Clean and return
    CUDA_CHECK(hipFree(d_clusters));
    cu_uf_destruct(d_uf);
    CUDA_CHECK(hipFree(d_edgecount_ptr));
    CUDA_CHECK(hipFree(d_edgelist));
    CUDA_CHECK(hipFree(d_offsets));
    CUDA_CHECK(hipFree(d_allseq));
    return ret;
}

std::vector<cu_alignment*> cu_local_align(const std::string& REF, const std::vector<fq_read*>& READS) {
    // Size consts
    int READ_LEN = READS.size();
    int THREADS = MAX_THREADS;
    int BLOCKS = (READ_LEN / MAX_THREADS) + 1;
    int DP_SIZE = (MAX_REF_LEN + 1) * (MAX_READ_LEN + 1) * READ_LEN;                // 200 * 170 * 4b * 25kseq ~= 3.4gb
    int CIGAR_SIZE = (MAX_CIGAR_LEN + 1) * READ_LEN;                                // 340 * 4b * 25kseq ~= 34m
    int ALIGN_LEN = READ_LEN * 3;

    // Host variables
    std::string temp = "";
    std::vector<uint32_t> h_offsets(READ_LEN);
    for(int i = 0; i < READ_LEN; ++i) {
        temp += READS[i]->get_seq();
        h_offsets[i] = temp.size() - 1;
    }
    int SEQ_LEN = temp.size() + 1;
    const char* h_allseq = temp.c_str();

    // Device variables
    char* d_allseq;
    char* d_cigarbuf;
    uint32_t* d_offsets;
    int* d_cache;
    int* d_align;

    // Allocate mem for device variables
    CUDA_CHECK(hipMalloc(&d_allseq, sizeof(char) * SEQ_LEN));
    CUDA_CHECK(hipMalloc(&d_cigarbuf, sizeof(char) * CIGAR_SIZE));
    CUDA_CHECK(hipMalloc(&d_offsets, sizeof(uint32_t) * READ_LEN));
    CUDA_CHECK(hipMalloc(&d_cache, sizeof(int) * DP_SIZE));
    CUDA_CHECK(hipMalloc(&d_align, sizeof(int) * ALIGN_LEN));
    
    // Copy mem host -> device/set mem
    CUDA_CHECK(hipMemset(d_cigarbuf, 0, sizeof(char) * CIGAR_SIZE));
    CUDA_CHECK(hipMemset(d_cache, 0, sizeof(int) * DP_SIZE));
    CUDA_CHECK(hipMemset(d_align, 0, sizeof(int) * ALIGN_LEN));
    CUDA_CHECK(hipMemcpy(d_allseq, h_allseq, sizeof(char) * SEQ_LEN, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, h_offsets.data(), sizeof(uint32_t) * READ_LEN, hipMemcpyHostToDevice));
    
    // Run local align kernel
    cu_local_alignment <<<BLOCKS, THREADS>>> (
        d_allseq,
        d_cigarbuf,
        d_cache,
        d_offsets,
        READ_LEN,
        d_align
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy mem back
    std::vector<int> h_align(ALIGN_LEN);
    std::vector<char> h_cigarbuf(CIGAR_SIZE);
    CUDA_CHECK(hipMemcpy(h_cigarbuf.data(), d_cigarbuf, sizeof(char) * CIGAR_SIZE, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_align.data(), d_align, sizeof(int) * ALIGN_LEN, hipMemcpyDeviceToHost));

    // Format data
    std::vector<cu_alignment*> ret;
    for(int i = 0; i < READ_LEN; ++i) {
        int H_ALIGN_BEGIN = i * 3;
        int CIGAR_BEGIN = (MAX_CIGAR_LEN + 1) * i;
        h_cigarbuf[CIGAR_BEGIN + MAX_CIGAR_LEN] = '\0';
        ret.push_back(new cu_alignment{
            new std::string(h_cigarbuf.data() + CIGAR_BEGIN),
            h_align[H_ALIGN_BEGIN],
            h_align[H_ALIGN_BEGIN + 1],
            h_align[H_ALIGN_BEGIN + 2]
        });
    }

    // Clean and return
    CUDA_CHECK(hipFree(d_align));
    CUDA_CHECK(hipFree(d_cache));
    CUDA_CHECK(hipFree(d_offsets));
    CUDA_CHECK(hipFree(d_cigarbuf));
    CUDA_CHECK(hipFree(d_allseq));
    return ret;
}