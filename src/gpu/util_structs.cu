#include "hip/hip_runtime.h"
#include "headers/util_structs.cuh"

#ifdef C_CU_UF

cu_union_find* cu_uf_construct(int n) {
    // Alloc host mem & device mem to copy over
    uint32_t* host_h = new uint32_t[n];
    uint32_t* host_p = new uint32_t[n];
    uint32_t* dev_h;
    uint32_t* dev_p;
    CUDA_CHECK(hipMalloc(&dev_h, sizeof(uint32_t) * n));
    CUDA_CHECK(hipMalloc(&dev_p, sizeof(uint32_t) * n));

    // Init values for union find
    for(int i = 0; i < n; ++i) {
        host_h[i] = 1;
        host_p[i] = i;
    }

    // Copy mem over
    CUDA_CHECK(hipMemcpy(dev_h, host_h, sizeof(uint32_t) * n, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_p, host_p, sizeof(uint32_t) * n, hipMemcpyHostToDevice));

    // Create host side struct
    cu_union_find host_uf;
    host_uf.h = dev_h;
    host_uf.p = dev_p;
    host_uf.LEN = n;

    // Copy host uf data over to device uf
    cu_union_find* dev_uf;
    CUDA_CHECK(hipMalloc(&dev_uf, sizeof(cu_union_find)));
    CUDA_CHECK(hipMemcpy(dev_uf, &host_uf, sizeof(cu_union_find), hipMemcpyHostToDevice));

    // Cleanup and return
    delete[] host_h;
    delete[] host_p; 
    return dev_uf;
}

void cu_uf_destruct(cu_union_find* d_uf) {
    // Copy to host to access p/h pointers
    cu_union_find h_uf;
    CUDA_CHECK(hipMemcpy(&h_uf, d_uf, sizeof(cu_union_find), hipMemcpyDeviceToHost));

    // Free device arrays
    CUDA_CHECK(hipFree(h_uf.h));
    CUDA_CHECK(hipFree(h_uf.p));

    // Free device uf struct
    CUDA_CHECK(hipFree(d_uf));
}

__device__ int __cu_uf_find(cu_union_find* UF, int x) {
    // Avoid recursion, do iteration even if it kinda sucks
    int root = x;
    while(true) {
        int parent = UF->p[root];
        if(parent == root) {
            break;
        }
        root = parent;
    }

    // Thread safety
    while(UF->p[x] != root) {
        int parent = UF->p[x];
        atomicCAS(&UF->p[x], parent, root);
        x = parent;
    }

    // Path compressed
    return root;
}

/*
 *  Modified join-by-height -> h[root] = count of all nodes in tree `NEED TO FIX`
 *  @param UF `cu_union_find*`
 *  @param x `int`
 *  @param y `int`
 *  @return `void`
 */
__device__ void __cu_uf_join(cu_union_find* UF, int x, int y) {
    int px = __cu_uf_find(UF, x);
    int py = __cu_uf_find(UF, y);
    if(px == py) {
        return;
    }
    
    // Need to use atomics for thread safety
    while(true) {
        px = __cu_uf_find(UF, px);
        py = __cu_uf_find(UF, py);
        
        // Joined
        if(px == py) {
            return;
        }
        
        // Get sizes
        uint32_t px_size = atomicAdd(&UF->h[px], 0);
        uint32_t py_size = atomicAdd(&UF->h[py], 0);

        // Join logic, favor x's parent as root in tiebreaker; favor lower index as root
        if(px_size >= py_size) {
            if(atomicCAS(&UF->p[py], py, px) == py) {
                // Linked py -> px, h[px] += h[py]
                atomicAdd(&UF->h[px], py_size);
                break;
            }
        } else {
            if(atomicCAS(&UF->p[px], px, py) == px) {
                // Linked px -> py, h[py] += h[px]
                atomicAdd(&UF->h[py], px_size);
                break;
            }
        }

    }
}

__device__ bool __cu_uf_con(cu_union_find* UF, int x, int y) {
    return __cu_uf_find(UF, x) == __cu_uf_find(UF, y);
}

#endif

#ifdef C_KMER_HASH_TABLE

__device__ uint64_t __kh_hash(uint64_t key) {
    key = (~key) + (key << 21);
    key = key ^ (key >> 24);
    key = (key + (key << 3)) + (key << 8);
    key = key ^ (key >> 14);
    key = (key + (key << 2)) + (key << 4);
    key = key ^ (key >> 28);
    key = key + (key << 31);
    return key;
}

#endif