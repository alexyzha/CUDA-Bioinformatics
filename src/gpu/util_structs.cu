#include "hip/hip_runtime.h"
#include "headers/util_structs.cuh"

#ifdef C_CU_UF

cu_union_find* cu_uf_construct(int n) {
    // Alloc host mem & device mem to copy over
    uint32_t* host_h = new uint32_t[n];
    uint32_t* host_p = new uint32_t[n];
    uint32_t* dev_h;
    uint32_t* dev_p;
    hipMalloc(&dev_h, sizeof(uint32_t) * n);
    hipMalloc(&dev_p, sizeof(uint32_t) * n);

    // Init values for union find
    for(int i = 0; i < n; ++i) {
        host_h[i] = 1;
        host_p[i] = i;
    }

    // Copy mem over
    hipMemcpy(dev_h, host_h, sizeof(uint32_t) * n, hipMemcpyHostToDevice);
    hipMemcpy(dev_p, host_p, sizeof(uint32_t) * n, hipMemcpyHostToDevice);

    // Create host side struct
    cu_union_find host_uf;
    host_uf.h = dev_h;
    host_uf.p = dev_p;
    host_uf.LEN = n;

    // Copy host uf data over to device uf
    cu_union_find* dev_uf;
    hipMalloc(&dev_uf, sizeof(cu_union_find));
    hipMemcpy(dev_uf, &host_uf, sizeof(cu_union_find), hipMemcpyHostToDevice);

    // Cleanup and return
    delete[] host_h;
    delete[] host_p; 
    return dev_uf;
}

__device__ int cu_uf_find(cu_union_find* UF, int x) {
    // Avoid recursion, do iteration even if it kinda sucks
    int root = x;
    while(true) {
        int parent = UF->p[root];
        if(parent == root) {
            break;
        }
        root = parent;
    }

    // Thread safety
    while(UF->p[x] != root) {
        int parent = UF->p[x];
        atomicCAS(&UF->p[x], parent, root);
        x = parent;
    }

    // Path compressed
    return root;
}

/*
 *  Modified join-by-height -> h[root] = count of all nodes in tree `NEED TO FIX`
 *  @param UF `cu_union_find*`
 *  @param x `int`
 *  @param y `int`
 *  @return `void`
 */
__device__ void cu_uf_join(cu_union_find* UF, int x, int y) {
    int px = cu_uf_find(UF, x);
    int py = cu_uf_find(UF, y);
    if(px == py) {
        return;
    }
    
    // Need to use atomics for thread safety
    while(true) {
        px = cu_uf_find(UF, px);
        py = cu_uf_find(UF, py);
        
        // Joined
        if(px == py) {
            return;
        }
        
        // Join logic, favor x's parent as root in tiebreaker; favor lower index as root
        if(px < py) {
            if(atomicCAS(&UF->p[py], py, px) == py) {
                // Equal height
                if(UF->h[px] == UF->h[py]) {
                    atomicAdd(&UF->h[px], 1);
                }
                return;
            }
        } else {
            if(atomicCAS(&UF->p[px], px, py) == px) {
                // Equal height
                if(UF->h[px] == UF->h[py]) {
                    atomicAdd(&UF->h[py], 1);
                }
                return;
            }
        }

    }
}

__device__ bool cu_uf_con(cu_union_find* UF, int x, int y) {
    return cu_uf_find(UF, x) == cu_uf_find(UF, y);
}

#endif

#ifdef C_KMER_HASH_TABLE

__device__ uint64_t kh_hash(uint64_t key) {
    key = (~key) + (key << 21);
    key = key ^ (key >> 24);
    key = (key + (key << 3)) + (key << 8);
    key = key ^ (key >> 14);
    key = (key + (key << 2)) + (key << 4);
    key = key ^ (key >> 28);
    key = key + (key << 31);
    return key;
}

#endif