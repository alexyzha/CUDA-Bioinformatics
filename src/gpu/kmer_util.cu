#include "hip/hip_runtime.h"
#include "headers/kmers.cuh"

__device__ void cu_count_kmers(kh_pair<uint64_t>* MAP, char* ALL_SEQ, size_t* OFFSETS, size_t K, size_t LEN, size_t MAP_LEN) {
    // OOB check for block/thread & K OOB check
    int SEQ_NUM = blockIdx.x * blockDim.x + threadIdx.x;
    if(SEQ_NUM >= LEN || K > K_MAX || K <= 0) {
        return;
    }

    // Rolling hash for O(1) kmer key creation
    uint64_t hash = 0;
    uint64_t mask = (1ULL << (K * 2)) - 1;
    size_t SEQ_BEGIN = OFFSETS[SEQ_NUM - 1] + 1;
    size_t SEQ_SIZE = OFFSETS[SEQ_NUM] + 1 - SEQ_BEGIN;
    if(SEQ_SIZE < K) {
        return;
    }

    // Rolling hash main logic
    for(int i = 0; i < SEQ_SIZE; ++i) {
        hash <<= 2;

        // __base_to_bit returns 0x80 in some cases. For ACGT bases, mask out non-2-bit values
        hash |= (static_cast<char>(0b11) & __base_to_bit(ALL_SEQ[SEQ_BEGIN + i]));

        // From index SEQ_BEGIN to SEQ_BEGIN + K - 1, hash is not yet a full kmer
        if(i >= K - 1) {
            // Get hash
            uint64_t index = kh_hash(hash) % MAP_LEN;

            // Linear probing
            for(int i = 0; i < MAP_LEN; ++i) {
                int cur = (index + i) % MAP_LEN;
                int prev = atomicCAS(
                    (ULL*)&MAP[cur].key, 
                    (ULL)EMPTY, 
                    (ULL)hash
                );
                if(prev == hash || prev == EMPTY) {
                    atomicAdd(
                        (ULL*)&MAP[cur].value, 
                        (ULL)1
                    );
                    break;
                }
            }
        }
    }
}

__device__ void cu_index_kmers(kh_pair<uint32_t[MAP_MAX_INDICES + 1]>* MAP, char* ALL_SEQ, size_t* OFFSETS, size_t K, size_t LEN, size_t MAP_LEN) {
    // OOB check for block/thread & K OOB check
    int SEQ_NUM = blockIdx.x * blockDim.x + threadIdx.x;
    if(SEQ_NUM >= LEN || K > K_MAX || K <= 0) {
        return;
    }

    // Rolling hash for O(1) kmer key creation
    uint64_t hash = 0;
    uint64_t mask = (1ULL << (K * 2)) - 1;
    size_t SEQ_BEGIN = OFFSETS[SEQ_NUM - 1] + 1;
    size_t SEQ_SIZE = OFFSETS[SEQ_NUM] + 1 - SEQ_BEGIN;
    if(SEQ_SIZE < K) {
        return;
    }

    // Rolling hash main logic
    for(int i = 0; i < SEQ_SIZE; ++i) {
        hash <<= 2;

        // __base_to_bit returns 0x80 in some cases. For ACGT bases, mask out non-2-bit values
        hash |= (static_cast<char>(0b11) & __base_to_bit(ALL_SEQ[SEQ_BEGIN + i]));

        // From index SEQ_BEGIN to SEQ_BEGIN + K - 1, hash is not yet a full kmer
        if(i >= K - 1) {
            // Get hash
            uint64_t index = kh_hash(hash) % MAP_LEN;

            // Linear probing
            for(int i = 0; i < MAP_LEN; ++i) {
                int cur = (index + i) % MAP_LEN;
                uint64_t prev = atomicCAS((ULL*)&MAP[cur].key, (ULL)EMPTY, (ULL)hash);

                // Insert new or found a match in map
                if(prev == EMPTY || prev == hash) {
                    // value[0] = last free index
                    uint32_t* values = &MAP[cur].value[0];
                    uint32_t count = atomicAdd(&values[0], 1);

                    // Prevent overflow/add index to list
                    if(count < MAP_MAX_INDICES) {
                        values[count + 1] = SEQ_NUM;
                    } else {
                        atomicSub(&values[0], 1);
                    }
                    break;
                }
            }
        }
    }
}

__device__ void cu_get_kmer_overlaps(kh_pair<uint32_t[MAP_MAX_INDICES + 1]>* MAP, size_t MAP_LEN, uint32_t* EDGE_LIST, uint32_t* EDGE_COUNT, uint32_t MAX_EDGES) {
    // OOB check for block/thread
    int SEQ_NUM = blockIdx.x * blockDim.x + threadIdx.x;
    if(SEQ_NUM >= MAP_LEN) {
        return;
    }

    // Get array of matching indices
    uint32_t* matches = MAP[SEQ_NUM].value;
    uint32_t count = matches[0];

    // Iterate through all pairs
    for(int i = 1; i <= count; ++i) {
        for(int j = i + 1; j <= count; ++j) {
            uint32_t src = matches[i];
            uint32_t dest = matches[j];

            // Add edges to list, watch for end of array
            int index = atomicAdd(EDGE_COUNT, 2);
            if(index + 1 >= MAX_EDGES) {
                return;
            }

            // Append pair to end
            EDGE_LIST[index] = src;
            EDGE_LIST[index + 1] = dest;
        }
    }
}

__device__ void cu_get_uf(cu_union_find* UF, size_t LEN, size_t NODES, uint32_t* EDGE_LIST, uint32_t EDGE_COUNT) {
    // Block/thread OOB checks
    int INDEX = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    if(INDEX + 1 >= EDGE_COUNT) {
        return;
    }

    // UF join
    uint32_t src = EDGE_LIST[INDEX];
    uint32_t dest = EDGE_LIST[INDEX + 1];
    cu_uf_join(UF, src, dest);
}

__device__ void cur_get_clusters{cu_union_find* UF, kh_pair<uint32_t[MAX_CLUSTER_SIZE + 1]> * MAP, size_t LEN, size_t MAP_LEN, size_t K} {
    // Block/thread OOB checks
    int INDEX = blockIdx.x * blockDim.x + threadIdx.x;
    if(INDEX >= LEN) {
        return;
    }

    // Check root size
    int px = cu_uf_find(UF, INDEX);
    if(UF->h[px] < K) {
        return;
    }

    // Add to cluster
    uint64_t index = kh_hash((ULL)INDEX) % MAP_LEN;
    for(int i = 0; i < MAP_LEN; ++i) {
        int cur = (index + i) % MAP_LEN;
        uint64_t prev = atomicCAS((ULL*)&MAP[cur].key, (ULL)EMPTY, (ULL)INDEX);

        // Insert new or found a match in map
        if(prev == EMPTY || prev == INDEX) {
            // cluster[0] = last free index
            uint32_t* cluster = &MAP[cur].value[0];
            uint32_t count = atomicAdd(&cluster[0], 1);

            // Prevent overflow/add index to list
            if(count < MAX_CLUSTER_SIZE) {
                cluster[count + 1] = INDEX;
            } else {
                atomicSub(&cluster[0], 1);
            }
            break;
        }
    }
}